/*
 *  Copyright (C) by Argonne National Laboratory.
 *      See COPYRIGHT in top-level directory.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MPL_gpu_kernel_trigger(volatile int *var)
{
    *var = 1;
    __threadfence_system();
}

__global__ void MPL_gpu_kernel_wait(volatile int *var)
{
    while(*var == 0);
}

extern "C"
void MPL_gpu_enqueue_trigger(volatile int *var, hipStream_t stream)
{
    hipError_t cerr;
    void *args[] = {&var};
    cerr = hipLaunchKernel(reinterpret_cast<const void*>((const void *) MPL_gpu_kernel_trigger), dim3(1,1,1), dim3(1,1,1),
                            args, 0, stream);
    if (cerr != hipSuccess) {
        fprintf(stderr, "CUDA Error (%s): %s\n", __func__, hipGetErrorString(cerr));
    }
}

extern "C"
void MPL_gpu_enqueue_wait(volatile int *var, hipStream_t stream)
{
    hipError_t cerr;
    void *args[] = {&var};
    cerr = hipLaunchKernel(reinterpret_cast<const void*>((const void *) MPL_gpu_kernel_wait), dim3(1,1,1), dim3(1,1,1),
                            args, 0, stream);
    if (cerr != hipSuccess) {
        fprintf(stderr, "CUDA Error (%s): %s\n", __func__, hipGetErrorString(cerr));
    }
}
