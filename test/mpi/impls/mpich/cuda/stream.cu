#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 */

#include <mpi.h>
#include <stdio.h>
#include <assert.h>

const int N = 1000000;
const int a = 2.0;

static void init_x(float *x)
{
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
    }
}

static void init_y(float *y)
{
    for (int i = 0; i < N; i++) {
        y[i] = 2.0f;
    }
}

static int check_result(float *y)
{
    float maxError = 0.0f;
    int errs = 0;
    for (int i = 0; i < N; i++) {
        if (abs(y[i] - 4.0f) > 0.01) {
            errs++;
            maxError = max(maxError, abs(y[i]-4.0f));
        }
    }
    if (errs > 0) {
        printf("%d errors, Max error: %f\n", errs, maxError);
    }
    return errs;
}

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
    int errs = 0;

    hipStream_t stream;
    hipStreamCreate(&stream);

    int mpi_errno;
    int rank, size;
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    if (size < 2) {
        printf("This test require 2 processes\n");
        exit(1);
    }

    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));

    if (rank == 0) {
        init_x(x);
    } else if (rank == 1) {
        init_y(y);
    }

    MPI_Info info;
    MPI_Info_create(&info);
    MPI_Info_set(info, "type", "hipStream_t");
    MPIX_Info_set_hex(info, "value", &stream, sizeof(stream));

    MPIX_Stream mpi_stream;
    MPIX_Stream_create(info, &mpi_stream);

    MPI_Info_free(&info);

    MPI_Comm stream_comm;
    MPIX_Stream_comm_create(MPI_COMM_WORLD, mpi_stream, &stream_comm);

    /* Rank 0 sends x data to Rank 1, Rank 1 performs a * x + y and checks result */
    if (rank == 0) {
        hipMemcpyAsync(d_x, x, N*sizeof(float), hipMemcpyHostToDevice, stream);

        mpi_errno = MPIX_Send_enqueue(d_x, N, MPI_FLOAT, 1, 0, stream_comm);
        assert(mpi_errno == MPI_SUCCESS);

        hipStreamSynchronize(stream);
    } else if (rank == 1) {
        hipMemcpyAsync(d_y, y, N*sizeof(float), hipMemcpyHostToDevice, stream);

        mpi_errno = MPIX_Recv_enqueue(d_x, N, MPI_FLOAT, 0, 0, stream_comm, MPI_STATUS_IGNORE);
        assert(mpi_errno == MPI_SUCCESS);

        saxpy<<<(N+255)/256, 256, 0, stream>>>(N, a, d_x, d_y);

        hipMemcpyAsync(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);
    }

    if (rank == 1) {
        int errs = check_result(y);
        if (errs == 0) {
            printf("No Errors\n");
        }
    }

    MPI_Comm_free(&stream_comm);
    MPIX_Stream_free(&mpi_stream);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    hipStreamDestroy(stream);
    MPI_Finalize();
    return errs;
}
